#include <iostream>
#include <stdio.h>

#include <hip/hip_runtime.h>

using namespace std;

__global__ void VecAdd(float* A, float* B, float* C)
{
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
}

int main()
{

    // Kernel invocation with N threads
    float *A, *B, *C;
    int N = 100;
    hipMalloc((void**)&A, N*N*sizeof(float));
    hipMalloc((void**)&B, N*N*sizeof(float));
    hipMalloc((void**)&C, N*N*sizeof(float));
    
    float *a = (float *)malloc(N*N*sizeof(float));
    float *b = (float *)malloc(N*N*sizeof(float));
    float *c = (float *)malloc(N*N*sizeof(float));

    hipMemcpy(A, a, N * N * sizeof(*A), hipMemcpyHostToDevice);
    hipMemcpy(B, b, N * N * sizeof(*B), hipMemcpyHostToDevice);

    VecAdd<<<1, N>>>(A, B, C);

    hipMemcpy(c, C, N * N * sizeof(*C), hipMemcpyHostToDevice);

    hipFree(A) ; hipFree(B) ; hipFree(C) ;

}