#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

using namespace std;

__global__ void MatAdd(float A[N][N], float B[N][N], float C[N][N])
{
    int i = threadIdx.x;
    int j = threadIdx.y;
    C[i][j] = A[i][j] + B[i][j];
};


int main() {
    float *A, *B, *C;
    int N = 100;

    hipMalloc((void**)&A, N*N*sizeof(float));
    hipMalloc((void**)&B, N*N*sizeof(float));
    hipMalloc((void**)&C, N*N*sizeof(float));
    
    float *a = (float *)malloc(N*N*sizeof(float));
    float *b = (float *)malloc(N*N*sizeof(float));
    float *c = (float *)malloc(N*N*sizeof(float));

    hipMemcpy(A, a, N * N * sizeof(*A), hipMemcpyHostToDevice);
    hipMemcpy(B, b, N * N * sizeof(*B), hipMemcpyHostToDevice);

    int numBlocks = 1;
    dim3 threadsPerBlock(N,N);
    MatAdd << numBlocks, threadsPerBlock >>> (A,B,C);
    hipMemcpy(c, C, N * N * sizeof(*C), hipMemcpyHostToDevice);

    hipFree(A) ; hipFree(B) ; hipFree(C);
    
}
